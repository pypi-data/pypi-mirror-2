#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hipfft/hipfft.h>
#include <cutil_inline.h>

#include <cudabuffer.h>
#include <batchfft.h>
#include <defines.h>


void runTest(int x, int y, int z)
{
	unsigned int timer = 0;
	cutilCheckError(cutCreateTimer(&timer));

	int batch = TEST_BUFFER_SIZE / (x * y * z * sizeof(complexType));
	if(batch == 0)
	{
		printf("Resulting buffer size is too big, test skipped\n");
		return;
	}

	bool use_batchfft;
	hipfftHandle cufft_plan;
	batchfftHandle batchfft_plan;
	CudaBuffer<complexType> idata(x * y * z * batch), odata(x * y * z * batch);

	printf("--- (%d, %d, %d), batch %d\n", x, y, z, batch);
	float gflop = 5.0e-9 * log2((float)(x * y * z)) * x * y * z * batch;

	if(y == 1 && z == 1)
		use_batchfft = false;
	else
		use_batchfft = true;

	// prepare plans
	if(use_batchfft)
		if(z != 1)
			cufftSafeCall(batchfftPlan3d(&batchfft_plan, z, y, x, PLAN_TYPE, batch));
		else
			cufftSafeCall(batchfftPlan2d(&batchfft_plan, y, x, PLAN_TYPE, batch));
	else
		cufftSafeCall(hipfftPlan1d(&cufft_plan, x, PLAN_TYPE, batch));

	// Warming up
	if(use_batchfft)
		cufftSafeCall(batchfftExecute(batchfft_plan, (complexType*)idata, (complexType*)odata, HIPFFT_FORWARD));
	else
		cufftSafeCall(executePlan(cufft_plan, (complexType*)idata, (complexType*)odata, HIPFFT_FORWARD));
	cutilSafeCall(hipDeviceSynchronize());

	// measure out of place time
	cutilCheckError(cutStartTimer(timer));
	for(int i = 0; i < NUMITER; i++)
		if(use_batchfft)
			cufftSafeCall(batchfftExecute(batchfft_plan, (complexType*)idata, (complexType*)odata, HIPFFT_FORWARD));
		else
			cufftSafeCall(executePlan(cufft_plan, (complexType*)idata, (complexType*)odata, HIPFFT_FORWARD));
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckError(cutStopTimer(timer));
	printf("Out-of-place time: %f ms (%f GFLOPS)\n",
	       cutGetTimerValue(timer) / NUMITER,
	       gflop / (cutGetTimerValue(timer) / NUMITER / 1000));

	cutilCheckError(cutResetTimer(timer));

	// measure inplace
	cutilCheckError(cutStartTimer(timer));
	for(int i = 0; i < NUMITER; i++)
		if(use_batchfft)
			cufftSafeCall(batchfftExecute(batchfft_plan, (complexType*)idata, (complexType*)idata, HIPFFT_FORWARD));
		else
			cufftSafeCall(executePlan(cufft_plan, (complexType*)idata, (complexType*)idata, HIPFFT_FORWARD));
	cutilSafeCall(hipDeviceSynchronize());
	cutilCheckError(cutStopTimer(timer));
	printf("Inplace time: %f ms (%f GFLOPS)\n",
	       cutGetTimerValue(timer) / NUMITER,
	       gflop / (cutGetTimerValue(timer) / NUMITER / 1000));

	cutilCheckError( cutDeleteTimer( timer));

	if(use_batchfft)
		batchfftDestroy(batchfft_plan);
	else
		hipfftDestroy(cufft_plan);
}

int main(int argc, char** argv)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
		cutilDeviceInit(argc, argv);
	else
		hipSetDevice( cutGetMaxGflopsDeviceId() );

	// 1D
	runTest(16, 1, 1);
	runTest(1024, 1, 1);
	runTest(8192, 1, 1);

	// 2D
	runTest(16, 16, 1);
	runTest(128, 128, 1);
	runTest(1024, 1024, 1);

	// 3D
	runTest(16, 16, 16);
	runTest(32, 32, 128);
	runTest(128, 128, 128);
}
