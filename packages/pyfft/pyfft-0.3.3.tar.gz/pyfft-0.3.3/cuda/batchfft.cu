#include <cutil_inline.h>
#include <assert.h>

#include "defines.h"
#include "batchfft.h"
#include "transpose.cuh"

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftFillPlan(batchfftHandle *plan, int nx, int ny, int nz, hipfftType type, int batch)
{
	if(type != PLAN_TYPE)
		return HIPFFT_INVALID_TYPE;

	if(nx % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(ny % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(nz != 1 && nz % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(nz == 1)
	{
		plan->dim = 2;

		// Swap nx and ny so they correspoind to the 2D CUFFT API.
		// 2D cufft expects them in the order for a declared C array:
		//
		// complexType array[nx][ny];
		// hipfftPlan2d(plan, nx, ny, type);
		//
		// even though ny would be considered the "x" array index for row-major
		// array storage.
		plan->ny = nx;
		plan->nx = ny;
		plan->nz = 1;
	}
	else
	{
		plan->dim = 3;

		// Swap dimensions, the reason is the same as for 2D case.
		plan->nx = nz;
		plan->ny = ny;
		plan->nz = nx;
	}

	plan->type = type;
	plan->batch = batch;

	hipfftResult ret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	cudaret = hipMalloc(&(plan->temp), plan->nx * plan->ny * plan->nz * plan->batch * sizeof(complexType));
	if(cudaret != hipSuccess)
		return HIPFFT_ALLOC_FAILED;

	ret = hipfftPlan1d(&(plan->xplan), plan->nx, plan->type, plan->ny * plan->nz * plan->batch);
	if(ret != HIPFFT_SUCCESS)
	{
		hipFree(plan->temp);
		plan->temp = NULL;
		return ret;
	}

	ret = hipfftPlan1d(&(plan->yplan), plan->ny, plan->type, plan->nx * plan->nz * plan->batch);
	if(ret != HIPFFT_SUCCESS)
	{
		hipFree(plan->temp);
		plan->temp = NULL;
		hipfftDestroy(plan->xplan);
		return ret;
	}

	if(plan->dim == 3)
	{
		ret = hipfftPlan1d(&(plan->zplan), plan->nz, plan->type, plan->nx * plan->ny * plan->batch);
		if(ret != HIPFFT_SUCCESS)
		{
			hipFree(plan->temp);
			plan->temp = NULL;
			hipfftDestroy(plan->xplan);
			hipfftDestroy(plan->yplan);
			return ret;
		}
	}

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftPlan2d(batchfftHandle* plan, int nx, int ny, hipfftType type, int batch)
{
	return batchfftFillPlan(plan, nx, ny, 1, type, batch);
}

hipfftResult batchfftPlan3d(batchfftHandle* plan, int nx, int ny, int nz, hipfftType type, int batch)
{
	return batchfftFillPlan(plan, nx, ny, nz, type, batch);
}

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftDestroy(batchfftHandle &plan)
{
	assert(plan.temp != NULL);

	hipfftDestroy(plan.xplan);
	hipfftDestroy(plan.yplan);
	if(plan.dim == 3)
		hipfftDestroy(plan.zplan);
	hipFree(plan.temp);
	plan.temp = NULL;

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftExecute2D(batchfftHandle &plan, complexType* idata, complexType* odata, int sign)
{
	hipfftResult cufftret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	// Transform rows
	cufftret = executePlan(plan.xplan, idata, odata, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	// Transpose
	cudaret = transpose(plan.temp, odata, plan.nx, plan.ny, plan.batch * plan.nz);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	// Transform columns
	cufftret = executePlan(plan.yplan, plan.temp, plan.temp, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	// Transpose back
	cudaret = transpose(odata, plan.temp, plan.ny, plan.nx, plan.batch * plan.nz);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftExecute3D(batchfftHandle &plan, complexType* idata, complexType* odata, int sign)
{
	hipfftResult cufftret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	cufftret = batchfftExecute2D(plan, idata, odata, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	cudaret = transpose(plan.temp, odata, plan.nx * plan.ny, plan.nz, plan.batch);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	cufftret = executePlan(plan.zplan, plan.temp, plan.temp, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	cudaret = transpose(odata, plan.temp, plan.nz, plan.nx * plan.ny, plan.batch);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	return HIPFFT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftExecute(batchfftHandle &plan, complexType* idata, complexType* odata, int sign)
{
	if(plan.dim == 2)
		return batchfftExecute2D(plan, idata, odata, sign);
	else if(plan.dim == 3)
		return batchfftExecute3D(plan, idata, odata, sign);
	else
		return HIPFFT_INVALID_PLAN;
}
