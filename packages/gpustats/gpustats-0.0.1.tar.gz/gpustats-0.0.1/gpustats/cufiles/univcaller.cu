#include "hip/hip_runtime.h"
/*
  Automatically-generated kernel for %(name)s

  For univariate distributions
 */

__global__ void k_%(name)s(float* output,
                           float* data,
                           float* params,
                           float* design) {

  // Think of a more elegant, efficient way of doing this
  // use shared memory?
  unsigned int data_per_block, params_per_block;
  unsigned int nobs, nparams, params_stride;

  // inelegant, perhaps...
  data_per_block = design[0];
  params_per_block = design[1];
  nobs = design[2];
  nparams = design[3];
  params_stride = design[4];

  unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

  unsigned int rel_param = tid / data_per_block;
  unsigned int rel_data = tid - rel_param * data_per_block;

  unsigned int obs_num = data_per_block * blockIdx.x + rel_data;
  unsigned int param_num = params_per_block * blockIdx.y + rel_param;
  unsigned int result_idx = nobs * param_num + obs_num;

  // set up shared data
  extern __shared__ float shared_data[];
  float* sh_params = shared_data;
  float* sh_data = sh_params + params_per_block * params_stride;
  float* sh_result = sh_data + data_per_block;

  copy_chunks(data + data_per_block * blockIdx.x,
              sh_data, tid,
              min(nobs - data_per_block * blockIdx.x,
                  data_per_block));

  copy_chunks(params + params_per_block * blockIdx.y * params_stride,
              sh_params, tid,
              min(params_per_block,
                  nparams - params_per_block * blockIdx.y) * params_stride);

  __syncthreads();

  // allocated enough shared memory so that this will not walk out of bounds
  // no matter what, though some of the results will be garbage
  sh_result[tid] = %(name)s(sh_data + rel_data,
                            sh_params + rel_param * params_stride);
  __syncthreads();

  // output is column-major, so this will then coalesce
  if (obs_num < nobs & param_num < nparams) {
	output[result_idx] = obs_num;
    output[result_idx] = sh_result[tid];
  }
}
